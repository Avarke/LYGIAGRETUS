#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>

using namespace std;



__device__ int calc_square(int val)
{
    return val * val;
}

__global__ void square(int *numbers, int arraySize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int a;
    for (int i = idx; i < arraySize; i += stride)
    {
        a = calc_square(numbers[i]);
        numbers[i] = a;
    }
}

int main()
{
    const int arraySize = 100;
    int num[arraySize];

    for (int i = 0; i < 100; i++)
    {
        num[i] = i;
    }
    
    int* d_num;




    hipMalloc((void**)&d_num,  arraySize * sizeof(int));
    hipMemcpy(d_num, num, arraySize * sizeof(int), hipMemcpyHostToDevice);
    square << <2, 5 >> > (d_num, arraySize);
    hipDeviceSynchronize();


    hipMemcpy(num, d_num, arraySize * sizeof(int), hipMemcpyDeviceToHost); 
    printf("Processed array:\n");
    for (int i = 0; i < arraySize; i++)
    {
        printf("num[%d] = %d\n", i, num[i]);
    }

    // Free device memory
    hipFree(d_num);

    return 0;
}

